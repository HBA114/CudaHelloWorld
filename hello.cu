#include <stdio.h>
#include <hip/hip_runtime.h>

// //! https://www.youtube.com/watch?v=cvo3gnInQ7M&ab_channel=JashKhatri

using namespace std;

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}


int main() {
    cuda_hello<<<1,1>>>();
    printf("Hello World from CPU!\n");
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}

/*

compile:    nvcc -arch compute_50 hello.cu
run:        ./a.out 

*/