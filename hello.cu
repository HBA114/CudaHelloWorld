// // #include "cuda_runtime.h"
// // #include "device_launch_parameters.h"

// #include <stdio.h>
// #include <cuda.h>


// //! https://www.youtube.com/watch?v=cvo3gnInQ7M&ab_channel=JashKhatri

// using namespace std;

// __global__ void cuda_hello(){
//     printf("Hello World from GPU!\n");
// }


// int main() {
//     cuda_hello<<<1,1>>>();
//     cuda_hello<<<1,1>>>();
//     printf("Hello World from CPU!\n");
//     cudaDeviceSynchronize();
//     cudaDeviceReset();
//     return 0;
// }



#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    print_kernel<<<10, 10>>>();
    hipDeviceSynchronize();
}

/*

hbasri@HP-ProBook:~/.../cuda-Projects/HelloWorld$ nvcc -arch compute_50 hello.cu
hbasri@HP-ProBook:~/.../cuda-Projects/HelloWorld$ ./a.out 

*/